#include <stdio.h>
#include <hip/hip_runtime.h>
#include <unistd.h>

#define N 10*1024*1024

__global__ void gpu_vector_add(int* a, int* b, int* c)
{
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

int main()
{
	//static int a[N], b[N], c[N];   /*Variáveis na memória principal*/
	int *a, *b, *c;
	int *d_a, *d_b, *d_c; /*Variáveis que alocaremos na memória da GPU*/
	int i;
	size_t size = N*sizeof(int);
	int err;

	a = (int*) malloc(size);
	b = (int*) malloc(size);
	c = (int*) malloc(size);

	err = hipMalloc((void **)&d_a, size); /*Aloque um inteiro na memória de vídeo e faça d_a apontar para ele.*/
	if (err != hipSuccess)
	{
		return 1;
	}
	err = hipMalloc((void **)&d_b, size);
	if (err != hipSuccess)
	{
		return 1;
	}
	err = hipMalloc((void **)&d_c, size);
	if (err != hipSuccess)
	{
		return 1;
	}

	for (i = 0; i < N; ++i)
		a[i] = 42;

	for (i = 0; i < N; ++i)
		b[i] = 1337;

	/*Copie a e b para os seus respectivos espaços alocados na GPU*/
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	                                       /* O que é isso?
											* a flag cudaMemcpyHostToDevice 
											* é parte de uma enum que especifica
											* o fluxo de dados. HostToDevice
											* especifica que copiaremos os
											* dados da RAM para a GRAM
											*/
	
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	
	gpu_vector_add<<<N,1>>>(d_a, d_b, d_c);
	       /*N = número de blocos. 1 bloco = um conjunto de threads*/

	usleep(1000000);
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	printf("Resultado: %d\n", c[0]);
	/*Libera a memória na placa. E se eu não liberar?*/
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	free(a); free(b); free(c);
	return 0;
}
