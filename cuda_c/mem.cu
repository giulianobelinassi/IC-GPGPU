#include <stdio.h>
#include <hip/hip_runtime.h>

int main()
{
	size_t m_free, m_total;
	double* array;
	
	hipMemGetInfo(&m_free, &m_total);
	hipMalloc((void **)&array, 20*1024*1024*sizeof(double));

	hipMemGetInfo(&m_free, &m_total);
	printf("Livre: %ld, Total %ld\n", m_free, m_total);

	return 0;
}
