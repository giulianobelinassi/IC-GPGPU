#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void gpu_add(int* a, int* b, int* c)
{
	*c = *a + *b;
}

int main()
{
	int a, b, c;          /*Variáveis na stack desse programa (na memória principal)*/
	int *d_a, *d_b, *d_c; /*Variáveis que alocaremos na memória da GPU*/

	hipMalloc((void **)&d_a, sizeof(int)); /*Aloque um inteiro na memória de vídeo e faça d_a apontar para ele.*/
	hipMalloc((void **)&d_b, sizeof(int));
	hipMalloc((void **)&d_c, sizeof(int));
	
	a = 42;
	b = 1337;

	/*Copie a e b para os seus respectivos espaços alocados na GPU*/
	hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
	                                       /* O que é isso?
											* a flag cudaMemcpyHostToDevice 
											* é parte de uma enum que especifica
											* o fluxo de dados. HostToDevice
											* especifica que copiaremos os
											* dados da RAM para a GRAM
											*/
	
	hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);
	
	gpu_add<<<1,1>>>(d_a, d_b, d_c);
	       /*pq eu preciso dessa template ainda é um mistério.*/

	hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);

	printf("Resultado: %d\n", c);
	/*Libera a memória na placa. E se eu não liberar?*/
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}
