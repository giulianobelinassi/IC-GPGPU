#include <stdio.h>
#include <hip/hip_runtime.h>
#include <unistd.h>

#define N 20*1024*1024
#define THREADS_PER_BLOCK 512

#ifdef __cplusplus
#define restrict __restrict__
#endif

__global__ void gpu_vector_add(int* a, int* b, int* c)
{
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	c[index] = a[index] + b[index];
}

void cpu_vector_add(int* restrict a, int* restrict b, int* restrict c)
{
	int i;

	for (i = 0; i < N; ++i)
		c[i] = a[i] + b[i];

}

int main()
{
	//static int a[N], b[N], c[N];   /*Variáveis na memória principal*/
	int *a, *b, *c;
	int *d_a, *d_b, *d_c; /*Variáveis que alocaremos na memória da GPU*/
	int i;
	size_t size = N*sizeof(int);
	int err;

	a = (int*) malloc(size);
	b = (int*) malloc(size);
	c = (int*) malloc(size);

	err = hipMalloc((void **)&d_a, size); /*Aloque um inteiro na memória de vídeo e faça d_a apontar para ele.*/
	if (err != hipSuccess)
	{
		return 1;
	}
	err = hipMalloc((void **)&d_b, size);
	if (err != hipSuccess)
	{
		return 1;
	}
	err = hipMalloc((void **)&d_c, size);
	if (err != hipSuccess)
	{
		return 1;
	}

	for (i = 0; i < N; ++i)
		a[i] = 42;

	for (i = 0; i < N; ++i)
		b[i] = 1337;

	/*Copie a e b para os seus respectivos espaços alocados na GPU*/
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	                                       /* O que é isso?
											* a flag cudaMemcpyHostToDevice 
											* é parte de uma enum que especifica
											* o fluxo de dados. HostToDevice
											* especifica que copiaremos os
											* dados da RAM para a GRAM
											*/
	
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	
	gpu_vector_add<<<N/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, d_c);
	       /*N = número de blocos. 1 bloco = um conjunto de threads*/

	hipDeviceSynchronize();
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	printf("Resultado: %d\n", c[0]);
	/*Libera a memória na placa. E se eu não liberar?*/
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	free(a); free(b); free(c);
	return 0;
}
